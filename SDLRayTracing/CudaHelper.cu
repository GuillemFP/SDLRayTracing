#include "CudaHelper.cuh"

#if USE_CUDA
#include "Math.h"
#include "Ray.h"
#include "Vector3.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include ""

int main()
{
	return 0;
}

namespace CudaHelper
{
	void CalculatePixelColorSamples(Vector3* dColors, const RayTracingConstants* constants, const Ray* rays, const Entity* entities, const size_t* numberOfEntities, math::LCG* randomGenerators)
	{
		CalculatePixelColorSamples<<<1,1>>>(_dColorSamples, _dConstants, _dRays, _dEntities, _dEntityNumber, _dRandomGenerators);
	}

	CUDA_KERNEL void CalculatePixelColorSamples(Vector3* dColors, const RayTracingConstants* constants, const Ray* rays, const Entity* entities, const size_t* numberOfEntities, math::LCG* randomGenerators)
	{
		int blockIndex = blockIdx.x;

		Vector3& colorResult = dColors[blockIndex];
		const Ray& ray = rays[blockIndex];
		math::LCG& randomGenerator = randomGenerators[blockIndex];
	}
}
#endif // USE_CUDA